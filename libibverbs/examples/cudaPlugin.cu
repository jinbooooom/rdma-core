#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

// 宏定义
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return -1; \
    } \
} while(0)

#define logd(fmt, ...) printf("[CUDA] " fmt "\n", ##__VA_ARGS__)

// 全局变量
static int g_initialized = 0;

/**
 * GPU kernel函数：触发门铃
 */
__global__ void trigger_doorbell_kernel(void *gpu_bf, void *gpu_ctrl) {
    // 触发门铃：将ctrl的值写入bf
    printf("kernel gpu_bf=%p, gpu_ctrl=%p\n", gpu_bf, gpu_ctrl);
    *((volatile uint64_t *)gpu_bf) = *(uint64_t *)gpu_ctrl;
}

/**
 * 初始化CUDA环境
 */
extern "C" int init_cuda() {
    if (g_initialized) {
        logd("CUDA already initialized");
        return 0;
    }
    
    logd("Initializing CUDA environment");
    
    // 设置CUDA设备（使用第一个可用设备）
    CUDA_CHECK(hipSetDevice(0));
    
    // 打印GPU信息
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    logd("Found %d CUDA devices", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        logd("Device %d: %s", i, prop.name);
    }
    
    g_initialized = 1;
    logd("CUDA environment initialized successfully");
    return 0;
}

/**
 * 清理CUDA环境
 */
extern "C" int cleanup_cuda() {
    logd("Cleaning up CUDA environment");
    g_initialized = 0;
    logd("CUDA environment cleaned up successfully");
    return 0;
}

/**
 * 将主机虚拟地址转换为GPU虚拟地址
 * @param hostVA 主机虚拟地址
 * @param size 内存大小
 * @param type 内存类型：0=bf, 1=ctrl
 * @param gpuVA 输出：GPU虚拟地址
 * @return 0成功，-1失败
 */
extern "C" int ConvertHostVA2GpuVA(void *hostVA, size_t size, int type, void **gpuVA) {
    if (!g_initialized) {
        fprintf(stderr, "CUDA not initialized\n");
        return -1;
    }
    
    if (!hostVA || !gpuVA) {
        fprintf(stderr, "Invalid parameters\n");
        return -1;
    }
    
    logd("Converting hostVA %p (size=%lu, type=%d) to GPU VA", hostVA, size, type);
    
    // 根据类型选择不同的标志
    auto flag = hipHostRegisterMapped; // 默认标志
    if (type == 0) { // bf类型
        flag = hipHostRegisterIoMemory | hipHostRegisterMapped;
        logd("Using bf flags: hipHostRegisterIoMemory | hipHostRegisterMapped");
    } else if (type == 1) { // ctrl类型
        flag = hipHostRegisterMapped;
        logd("Using ctrl flags: hipHostRegisterMapped");
    }
    
    CUDA_CHECK(hipHostRegister(hostVA, size, flag));
    
    // 获取GPU设备指针
    CUDA_CHECK(hipHostGetDevicePointer(gpuVA, hostVA, 0));
    
    logd("HostVA %p -> GPU VA %p", hostVA, *gpuVA);
    return 0;
}

/**
 * 在GPU上触发门铃
 * @param gpu_bf GPU端的bf指针
 * @param gpu_ctrl GPU端的ctrl指针
 * @return 0成功，-1失败
 */
extern "C" int TriggerDoorbell(void *gpu_bf, void *gpu_ctrl) {
    if (!g_initialized) {
        fprintf(stderr, "CUDA not initialized\n");
        return -1;
    }
    
    if (!gpu_bf || !gpu_ctrl) {
        fprintf(stderr, "Invalid GPU pointers\n");
        return -1;
    }
    
    logd("Triggering doorbell on GPU: bf=%p, ctrl=%p", gpu_bf, gpu_ctrl);
    
    // 启动GPU kernel
    trigger_doorbell_kernel<<<1, 1>>>(gpu_bf, gpu_ctrl);
    
    // 检查kernel执行是否成功
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    logd("GPU doorbell triggered successfully");
    return 0;
}

/**
 * 取消注册主机内存
 * @param hostVA 主机虚拟地址
 * @return 0成功，-1失败
 */
extern "C" int UnregisterHostVA(void *hostVA) {
    if (!g_initialized) {
        fprintf(stderr, "CUDA not initialized\n");
        return -1;
    }
    
    if (!hostVA) {
        fprintf(stderr, "Invalid hostVA\n");
        return -1;
    }
    
    logd("Unregistering hostVA %p", hostVA);
    CUDA_CHECK(hipHostUnregister(hostVA));
    logd("HostVA %p unregistered successfully", hostVA);
    return 0;
}
